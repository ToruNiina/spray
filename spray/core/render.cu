#include "hip/hip_runtime.h"
#include <spray/core/cuda_assert.hpp>
#include <spray/core/show_image.cuh>
#include <spray/core/color.hpp>
#include <spray/core/material.hpp>
#include <spray/core/world.cuh>
#include <spray/geom/sphere.hpp>
#include <spray/geom/ray.hpp>
#include <spray/geom/collide.hpp>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sort.h>
#include <thrust/pair.h>

#include <hip/hip_vector_types.h>
#include <vector_functions.h>

namespace spray
{
namespace core
{

__device__
float fclampf(float x, float minimum, float maximum)
{
    return fminf(fmaxf(x, minimum), maximum);
}

__device__
uchar4 make_pixel(spray::core::color col)
{
    uchar4 pixel;
    pixel.x = std::uint8_t(fclampf(sqrtf(spray::core::R(col)) * 256, 0, 255));
    pixel.y = std::uint8_t(fclampf(sqrtf(spray::core::G(col)) * 256, 0, 255));
    pixel.z = std::uint8_t(fclampf(sqrtf(spray::core::B(col)) * 256, 0, 255));
    pixel.w = 0xFF;
    return pixel;
}

__global__
void render_kernel(const std::size_t width, const std::size_t height,
        const float rwidth, const float rheight,
        const spray::geom::point location,
        const spray::geom::point lower_left,
        const spray::geom::point horizontal,
        const spray::geom::point vertical,
        const std::size_t        N,
        thrust::device_ptr<const spray::core::material> material,
        thrust::device_ptr<const spray::geom::sphere>   spheres,
        thrust::device_ptr<uchar4> img,
        thrust::device_ptr<std::uint32_t> first_hit_obj)
{
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;

    if(x >= width)  {return;}
    if(y >= height) {return;}
    const std::size_t offset = x + y * width;

    const spray::geom::point dst = lower_left +
                                   ((x+0.5f) *  rwidth) * horizontal +
                                   ((y+0.5f) * rheight) * vertical;
    const spray::geom::ray ray = spray::geom::make_ray(location, dst - location);

    std::uint32_t index = 0xFFFFFFFF;
    spray::geom::collision col;
    col.t = spray::geom::inf();
    for(std::size_t i=0; i<N; ++i)
    {
        const spray::geom::collision c = collide(ray, spheres[i], 0.0f);
        if(!isinf(c.t) && c.t < col.t)
        {
            index = i;
            col   = c;
        }
    }
    uchar4 pixel;
    if(index == 0xFFFFFFFF)
    {
        pixel.x = 0x00;
        pixel.y = 0x00;
        pixel.z = 0x00;
        pixel.w = 0x00;
    }
    else
    {
        const spray::core::material mat = material[index];
        const spray::core::color  color = mat.albedo * fabsf(spray::geom::dot(
                spray::geom::direction(ray), spray::geom::normal(col)));

        pixel = make_pixel(color);
    }
    img[offset] = pixel;
    first_hit_obj[offset] = index;
    return;
}

} // core
} // spray
