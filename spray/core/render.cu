#include "hip/hip_runtime.h"
#include <spray/core/cuda_assert.hpp>
#include <spray/core/show_image.cuh>
#include <spray/core/color.cuh>
#include <spray/core/material.hpp>
#include <spray/core/world.cuh>
#include <spray/geom/sphere.hpp>
#include <spray/geom/ray.hpp>
#include <spray/geom/collide.hpp>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>

namespace spray
{
namespace core
{

__global__
void render_kernel(const std::size_t width, const std::size_t height,
        const float rwidth, const float rheight,
        const spray::geom::point location,
        const spray::geom::point lower_left,
        const spray::geom::point horizontal,
        const spray::geom::point vertical,
        const std::size_t        N,
        thrust::device_ptr<const spray::core::material> material,
        thrust::device_ptr<const spray::geom::sphere>   spheres,
        thrust::device_ptr<uchar4> img,
        thrust::device_ptr<std::uint32_t> first_hit_obj)
{
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    if(x >= width || y >= height) {return;}

    const std::size_t offset = x + y * width;

    const spray::geom::point dst = lower_left +
                                   ((x+0.5f) *  rwidth) * horizontal +
                                   ((y+0.5f) * rheight) * vertical;
    const spray::geom::ray ray = spray::geom::make_ray(location, dst - location);

    std::uint32_t index = 0xFFFFFFFF;
    spray::geom::collision col;
    col.t = spray::geom::inf();
    for(std::size_t i=0; i<N; ++i)
    {
        const spray::geom::collision c = collide(ray, spheres[i], 0.0f);
        if(!isinf(c.t) && c.t < col.t)
        {
            index = i;
            col   = c;
        }
    }
    uchar4 pixel;
    if(index == 0xFFFFFFFF)
    {
        pixel.x = 0x00;
        pixel.y = 0x00;
        pixel.z = 0x00;
        pixel.w = 0x00;
    }
    else
    {
        const spray::core::material mat = material[index];
        const spray::core::color  color = mat.albedo * fabsf(spray::geom::dot(
                spray::geom::direction(ray), spray::geom::normal(col)));

        pixel = make_pixel(color);
    }
    img[offset] = pixel;
    first_hit_obj[offset] = index;
    return;
}

} // core
} // spray
