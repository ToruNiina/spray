#include "hip/hip_runtime.h"
#include <spray/core/pinhole_camera.cuh>
#include <spray/core/render.cuh>
#include <spray/core/show_image.cuh>
#include <spray/core/world.cuh>
#include <imgui.h>
#include <png++/png.hpp>

namespace spray
{
namespace core
{

std::unique_ptr<camera_base> make_pinhole_camera(
        std::string        name,
        spray::geom::point location,
        spray::geom::point direction,
        spray::geom::point view_up,
        float              fov,
        std::size_t        width,
        std::size_t        height
    )
{
    return std::make_unique<pinhole_camera>(std::move(name),
            location, direction, view_up, fov, width, height);
}

void pinhole_camera::reset(spray::geom::point location,
                           spray::geom::point direction,
                           spray::geom::point view_up,
                           float              fov,
                           std::size_t        width,
                           std::size_t        height)
{
    this->width_   = width;
    this->height_  = height;
    this->rwidth_  = 1.0f / width;
    this->rheight_ = 1.0f / height;

    const float aspect_ratio = static_cast<float>(width) / height;
    const float theta        = fov * 3.14159265 / 180.0;
    const float half_height  = std::tan(theta * 0.5);
    const float half_width   = half_height * aspect_ratio;

    const auto w = -spray::geom::unit(direction);
    const auto u =  spray::geom::unit(spray::geom::cross(view_up, w));
    const auto v =  spray::geom::cross(w, u);

    this->field_of_view_ = fov;
    this->location_   = location;
    this->direction_  = -w;
    this->view_up_    = v;
    this->pitch_axis_ = u;
    this->lower_left_ = location - half_width * u - half_height * v - w;
    this->horizontal_ = (2 * half_width)  * u;
    this->vertical_   = (2 * half_height) * v;

    if(this->scene_.size() != width * height)
    {
        this->scene_.resize(width * height);
        this->host_first_hit_obj_.resize(width * height);
        this->device_first_hit_obj_.resize(width * height);
    }

    this->field_of_view_buf_ = this->field_of_view_;
    this->pos_buf_[0] = spray::geom::X(this->location_);
    this->pos_buf_[1] = spray::geom::Y(this->location_);
    this->pos_buf_[2] = spray::geom::Z(this->location_);
    this->dir_buf_[0] = spray::geom::X(this->direction_);
    this->dir_buf_[1] = spray::geom::Y(this->direction_);
    this->dir_buf_[2] = spray::geom::Z(this->direction_);
    this->vup_buf_[0] = spray::geom::X(this->view_up_);
    this->vup_buf_[1] = spray::geom::Y(this->view_up_);
    this->vup_buf_[2] = spray::geom::Z(this->view_up_);
    return ;
}

bool pinhole_camera::update_gui()
{
    ImGui::Begin(this->name_.c_str());
    const bool focused = !(ImGui::IsWindowFocused());

    ImGui::InputFloat ("View angle", std::addressof(this->field_of_view_buf_));
    ImGui::InputFloat3("Camera position",  pos_buf_.data());
    ImGui::InputFloat3("Camera direction", dir_buf_.data());
    ImGui::InputFloat3("Camera view-up",   vup_buf_.data());
    if(ImGui::Button("Apply changes"))
    {
        this->reset(
            spray::geom::make_point(pos_buf_[0], pos_buf_[1], pos_buf_[2]),
            spray::geom::make_point(dir_buf_[0], dir_buf_[1], dir_buf_[2]),
            spray::geom::make_point(vup_buf_[0], vup_buf_[1], vup_buf_[2]),
            this->field_of_view_buf_, this->width_, this->height_);
    }

    ImGui::InputText("File name", filename_buf_.data(), 256);
    if(ImGui::Button("Save image as png"))
    {
        thrust::host_vector<uchar4> pixels = this->scene_;
        assert(pixels.size() == this->width_ * this->height_);

        png::image<png::rgba_pixel> img(this->width_, this->height_);
        for(std::size_t y=0; y<this->height_; ++y)
        {
            for(std::size_t x=0; x<this->width_; ++x)
            {
                const auto pix = pixels[x + y * this->width_];
                img[y][x] = png::rgba_pixel(pix.x, pix.y, pix.z, pix.w);
            }
        }
        img.write(filename_buf_.data());
    }

    const auto framerate = ImGui::GetIO().Framerate;
    ImGui::Text("Application average %.3f ms/frame (%.1f FPS)",
                1000.0f / framerate, framerate);
    ImGui::End();
    return focused;
}

void pinhole_camera::render(
        const dim3 blocks, const dim3 threads, const hipStream_t stream,
        const world_base& wld_base, const buffer_array& bufarray)
{
    const auto& wld = dynamic_cast<spray::core::world const&>(wld_base);
    if(!wld.is_loaded())
    {
        wld.load();
    }

    spray::core::render_kernel<<<blocks, threads, 0, stream>>>(
        this->width_, this->height_, this->rwidth_, this->rheight_,
        this->location_, this->lower_left_, this->horizontal_, this->vertical_,
        wld.device_spheres().size(),
        thrust::device_pointer_cast(wld.device_materials().data()),
        thrust::device_pointer_cast(wld.device_spheres().data()),
        thrust::device_pointer_cast(this->scene_.data()),
        thrust::device_pointer_cast(this->device_first_hit_obj_.data())
        );
    this->host_first_hit_obj_ = device_first_hit_obj_;

    spray::core::show_image(
           blocks, threads, stream, bufarray.array(), this->width_, this->height_,
           thrust::device_pointer_cast(this->scene_.data()));
    return;
}

} // core
} // spray
